#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

extern "C" {
#include "gaussianSeparable_kernel.h"
}

#define BLOCKSIZE 32
#define MAX_LW 257 // CONSTRAINT: max size of lw (window) is 256
__device__ __constant__ double d_weights[MAX_LW];


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void copy3d(double *to, double *from, size_t cube_z, size_t cube_y, size_t cube_x) {
    #pragma omp parallel for
    for (size_t z = 0; z < cube_z; z++) {
        for (size_t y = 0; y < cube_y; y++) {
            for (size_t x = 0; x < cube_x; x++) {
                size_t cube_idx = (z * cube_y * cube_x) + (y * cube_x) + x;
                to[cube_idx] = from[cube_idx];
            }
        }
    }
}


void print_memory_use() {
    size_t free_byte ;
    size_t total_byte ;
    hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
    if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
    }
    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;
    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
    used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}




extern "C" double* init_out_cube(size_t cube_z, size_t cube_y, size_t cube_x) {
    double* out_cube;
    gpuErrchk( hipHostMalloc(&out_cube, sizeof(double) * cube_x * cube_y * cube_z) );
    return out_cube;
}







__device__ double convolve_1d_gpu_kernel_optimised(double *subcube, int sidx, int lw) {

    double sum = d_weights[lw] * subcube[sidx];
    double lo_val, hi_val, weight;

    for (int i = 1; i <= lw; i++) {
        weight = d_weights[lw + i];
        lo_val = subcube[sidx - i];
        hi_val = subcube[sidx + i];
        sum += weight * (lo_val + hi_val);
    }

    return sum;
}







__global__ void gaussian_filter_1d_gpu_kernel_optimised(double *d_in_cube, double *d_out_cube, size_t allocate_px, int lw, int stride, int cube_y, int cube_x) {

    int tx, ty, sx, sy, x, y, sidx, sstride, cube_idx, t, ct, end;
    extern __shared__ double subcube[];

    tx = threadIdx.x;
    ty = threadIdx.y;
    sstride = BLOCKSIZE + 2 * lw;
    x = blockDim.x * blockIdx.x + tx;
    y = blockDim.y * blockIdx.y + ty;
    cube_idx = (blockIdx.z * cube_x * cube_y) + (y * cube_x) + x;

    // if stride == cube_x then we are filtering along y. so load the data into shared memory contiguously for processing
    if (stride == cube_x) {
        t = ty;
        ct = y;
        end = cube_y;
        sx = tx;
        sy = lw + ty;
        sidx = sx * sstride + sy;
    } else {
        t = tx;
        ct = x;
        end = cube_x;
        sx = lw + tx;
        sy = ty;
        sidx = sy * sstride + sx;
    }

    // copy pixel into subcube shared memory
    ////////////////////////////////////////

    if (x >= cube_x || y >= cube_y) {
        subcube[sidx] = 0;
    } else {
        subcube[sidx] = d_in_cube[cube_idx];
    }

    // copy window into shared memory
    ////////////////////////////////////

    // padding
    if (t == 0) { // left boundary
        if (ct == 0) { // cube hard boundary, pad with zero
            for (int i = 1; i <= lw; i++) {
                subcube[sidx - i] = 0;
            }
        } else { // soft boundary, pad from cube pixels
            for (int i = 1; i <= lw; i++) {
                subcube[sidx - i] = d_in_cube[cube_idx - i * stride];
            }
        }

    } else if (t == BLOCKSIZE - 1) { // right boundary
        if (ct == end - 1) { // cube hard boundary, pad with zero
            for (int i = 1; i <= lw; i++) {
                subcube[sidx + i] = 0;
            }
        } else { // soft boundary, pad from cube pixels
            for (int i = 1; i <= lw; i++) {
                if (ct + i < end) { // the window column might be thicker than the remaining columns in the cube
                    subcube[sidx + i] = d_in_cube[cube_idx + i * stride];
                } else {
                    subcube[sidx + i] = 0;
                }
            }
        }
    }

    __syncthreads();

    // perform convolution
    //////////////////////

    if (x < cube_x && y < cube_y && cube_idx < allocate_px) {
        d_out_cube[cube_idx] = convolve_1d_gpu_kernel_optimised(subcube, sidx, lw);
    }

}

__device__ double convolve_1d_gpu_kernel(double *subcube, int sidx, int lw, int sstride) {

    double sum = d_weights[lw] * subcube[sidx];
    double lo_val, hi_val, weight;

    for (int i = 1; i <= lw; i++) {
        weight = d_weights[lw + i];
        lo_val = subcube[sidx - i * sstride];
        hi_val = subcube[sidx + i * sstride];
        sum += weight * (lo_val + hi_val);
    }

    return sum;
}

__global__ void gaussian_filter_1d_gpu_kernel(double *d_in_cube, double *d_out_cube, size_t allocate_px, int lw, int stride, int cube_y, int cube_x) {

    int tx, ty, sx, sy, x, y, sidx, sstride, cube_idx;
    extern __shared__ double subcube[];

    tx = threadIdx.x;
    ty = threadIdx.y;
    sx = lw + tx;
    sy = lw + ty;
    sstride = blockDim.x + 2 * lw;
    sidx = sy * sstride + sx;
    x = blockDim.x * blockIdx.x + tx;
    y = blockDim.y * blockIdx.y + ty;
    cube_idx = (blockIdx.z * cube_x * cube_y) + (y * cube_x) + x;

    // copy pixel into subcube shared memory
    ////////////////////////////////////////

    if (x >= cube_x || y >= cube_y) {
        subcube[sidx] = 0;
    } else {
        subcube[sidx] = d_in_cube[cube_idx];
    }

    // copy window into shared memory
    ////////////////////////////////////

    // X padding

    if (tx == 0) { // left boundary
        if (x == 0) { // cube hard boundary, pad with zero
            for (int i = 1; i <= lw; i++) {
                subcube[sidx - i] = 0;
            }
        } else { // soft boundary, pad from cube pixels
            for (int i = 1; i <= lw; i++) {
                subcube[sidx - i] = d_in_cube[cube_idx - i];
            }
        }

    } else if (tx == blockDim.x - 1) { // right boundary
        if (x == cube_x - 1) { // cube hard boundary, pad with zero
            for (int i = 1; i <= lw; i++) {
                subcube[sidx + i] = 0;
            }
        } else { // soft boundary, pad from cube pixels
            for (int i = 1; i <= lw; i++) {
                if (x + i < cube_x) { // the window column might be thicker than the remaining columns in the cube
                    subcube[sidx + i] = d_in_cube[cube_idx + i];
                } else {
                    subcube[sidx + i] = 0;
                }
            }
        }
    }

    // Y padding

    if (ty == 0) { // top boundary
        if (y == 0) { // cube hard boundary, pad with zero
            for (int i = 1; i <= lw; i++) {
                subcube[sidx - i * sstride] = 0;
            }
        } else { // soft boundary, pad from cube pixels
            for (int i = 1; i <= lw; i++) {
                subcube[sidx - i * sstride] = d_in_cube[cube_idx - i * stride];
            }
        }

    } else if (ty == blockDim.y - 1) { // bottom boundary
        if (y == cube_y - 1) { // cube hard boundary, pad with zero
            for (int i = 1; i <= lw; i++) {
                subcube[sidx + i * sstride] = 0;
            }
        } else { // soft boundary, pad from cube pixels
            for (int i = 1; i <= lw; i++) {
                if (y + i < cube_y) { // the window row might be thicker than the remaining rows in the cube
                    subcube[sidx + i * sstride] = d_in_cube[cube_idx + i * stride];
                } else {
                    subcube[sidx + i * sstride] = 0;
                }
            }
        }
    }


    __syncthreads();


    // perform convolution
    //////////////////////

    if (x < cube_x && y < cube_y && cube_idx < allocate_px) {
        if (stride == 1) { // need to differentiate between x and y convolution
            d_out_cube[cube_idx] = convolve_1d_gpu_kernel(subcube, sidx, lw, 1);
        } else {
            d_out_cube[cube_idx] = convolve_1d_gpu_kernel(subcube, sidx, lw, sstride);
        }
    }

}

double* generate_weights(double sd, int lw) {
    double *h_weights    = (double *) malloc(sizeof(double) * MAX_LW);
    h_weights[lw] = 1.0;
    double sum = 1.0;
    // generate gaussian row weights
    for (int i = 1; i <= lw; i++) {
        double tmp = exp(-0.5 * ((double) i * i) / (sd * sd));
        h_weights[lw + i] = tmp;
        h_weights[lw - i] = tmp;
        sum += 2.0 * tmp;
    }
    for (int i = 0; i <= 2 * lw; i++) {
        h_weights[i] /= sum;
    }
    return h_weights;
}


int get_available_out_bytes() {
    size_t free_byte, total_byte;
    gpuErrchk( hipMemGetInfo(&free_byte, &total_byte) );
    double free_db = (double)free_byte;
    return (int) (free_db - 1024.0 * 1024.0 * 15) / 2.0; // shave off 15MB so we don't crash by not being able to allocate the last bit of memory on the device
}

void gaussian_filter_1d(double *in, double *out, int cube_z, int cube_y, int cube_x, int ks, int stride) {

    // initialize gaussian filter weights
    double sd = ((double) ks) / 2.355;
    int lw = (int) (sd * 4 + 0.5);
    // note: the unoptimised kernel cannot run with the optimised sm_bytes setting, I don't think.
    // int sm_bytes        = (BLOCKSIZE + 2 * lw) * (BLOCKSIZE + 2 * lw) * sizeof(double);
    int sm_bytes_optimised = (BLOCKSIZE) * (BLOCKSIZE + 2 * lw) * sizeof(double);
    double* h_weights = generate_weights(sd, lw);

    // initialize size of cube to filter
    double *h_in = in;
    size_t plane_px = cube_y * cube_x;
    size_t total_cube_px = plane_px * cube_z;
    long total_cube_bytes = total_cube_px * sizeof(double);

    // determine how much memory is available for allocation on the device
    // and calculate how much z_height of the cube can fit into it
    size_t out_bytes = get_available_out_bytes();
    size_t out_px = out_bytes / sizeof(double);
    size_t z_height;
    if (total_cube_px > out_px) {
        z_height = (out_px - out_px % plane_px) / plane_px;
    } else {
        z_height = total_cube_px / plane_px;
    }
    size_t allocate_px = z_height * plane_px;
    size_t allocate_bytes = allocate_px * sizeof(double);

    // memory allocation
    double *d_in  = NULL;
    double *d_out = NULL;
    gpuErrchk( hipMalloc(&d_in,  allocate_bytes) );
    gpuErrchk( hipMalloc(&d_out, allocate_bytes) );

    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_weights), h_weights, MAX_LW * sizeof(double)) );
    gpuErrchk( hipHostRegister(in, total_cube_bytes, 0) );


    // process the cube in parts on the GPU
    size_t offset = 0;
    for (; offset < total_cube_px; offset += allocate_px) {

        if (offset + allocate_px > total_cube_px) {
            z_height = (total_cube_px - offset) / plane_px;
            allocate_px = z_height * plane_px;
            allocate_bytes = allocate_px * sizeof(double);
        }

        gpuErrchk( hipMemcpy(d_in, &h_in[offset], allocate_bytes, hipMemcpyHostToDevice) );
        dim3 dimBlock = dim3(BLOCKSIZE, BLOCKSIZE, 1);
        dim3 dimGrid = dim3(
            ceil( ((int) cube_x)   / (float) dimBlock.x),
            ceil( ((int) cube_y)   / (float) dimBlock.y),
            ceil( ((int) z_height) / (float) dimBlock.z)
        );
        // gaussian_filter_1d_gpu_kernel<<<dimGrid, dimBlock, sm_bytes>>>(d_in, d_out, allocate_px, lw, stride, cube_y, cube_x);
        gaussian_filter_1d_gpu_kernel_optimised<<<dimGrid, dimBlock, sm_bytes_optimised>>>(d_in, d_out, allocate_px, lw, stride, cube_y, cube_x);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipMemcpy(&h_in[offset], d_out, allocate_bytes, hipMemcpyDeviceToHost) );

    }

    // clean up
    hipHostUnregister(in);
    hipFree(d_in);
    hipFree(d_out);
    free(h_weights);

}



void gaussian_filter_GPU(double *in_cube, double *out_cube, int cube_z, int cube_y, int cube_x, int ky, int kx) {

    if (cube_z == 0 && cube_y == 0 && cube_x == 0) return;

    int stride;

    if (ky > 0) {
        stride = cube_x;
        gaussian_filter_1d(in_cube, out_cube, cube_z, cube_y, cube_x, ky, stride);
        // copy3d(in_cube, out_cube, cube_z, cube_y, cube_x);
    }

    if (kx > 0) {
        stride = 1;
        gaussian_filter_1d(in_cube, out_cube, cube_z, cube_y, cube_x, kx, stride);
        // copy3d(in_cube, out_cube, cube_z, cube_y, cube_x);
    }

}

